#include "hip/hip_runtime.h"

# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <time.h>
# include <string.h>
	
#if defined(_OPENMP)
# include <omp.h>
#endif

# include "poisson.cuh"
# include "../main.h"
# include "../timer.h"

#ifdef APP_JACOBI

double r8mat_rms(int nx, int ny, double *a_);
void rhs(int nx, int ny, double *f_, int block_size);
void timestamp(void);
double u_exact(double x, double y);
double uxxyy_exact(double x, double y);

/*
Purpose:

MAIN is the main program for POISSON_OPENMP.

Discussion:

POISSON_OPENMP is a program for solving the Poisson problem.

This program uses OpenMP for parallel execution.

The Poisson equation

- DEL^2 U(X,Y) = F(X,Y)

is solved on the unit square [0,1] x [0,1] using a grid of NX by
NX evenly spaced points.  The first and last points in each direction
are boundary points.

The boundary conditions and F are set so that the exact solution is

U(x,y) = sin ( pi * x * y)

so that

- DEL^2 U(x,y) = pi^2 * ( x^2 + y^2) * sin ( pi * x * y)

The Jacobi iteration is repeatedly applied until convergence is detected.

For convenience in writing the discretized equations, we assume that NX = NY.

Licensing:

This code is distributed under the GNU LGPL license.

Modified:

14 December 2011

Author:

John Burkardt
*/

/******************************************************************************/
double run(struct user_parameters* params)
{
	int matrix_size = params->matrix_size;
	if (matrix_size <= 0) {
		matrix_size = 512;
		params->matrix_size = matrix_size;
	}
	int block_size = params->blocksize;
	if (block_size <= 0) {
		block_size = 128;
		params->blocksize = block_size;
	}
	int niter = params->titer;
	if (niter <= 0) {
		niter = 4;
		params->titer = niter;
	}
	double dx;
	double dy;
	double error;
	int ii,i;
	int jj,j;
	int nx = matrix_size;
	int ny = matrix_size;
	double *f_ = (double *)malloc(nx * ny * sizeof(double));
	double (*f)[nx][ny] = (double (*)[nx][ny])f_;
	double *u_ = (double *)malloc(nx * ny * sizeof(double));
	double *unew_ = (double *)malloc(nx * ny * sizeof(double));
	memset(unew_,0,nx * ny * sizeof(double));
	memset(u_,0,nx * ny * sizeof(double));
	double (*unew)[nx][ny] = (double (*)[nx][ny])unew_;

	/* test if valid */
	if ( (nx % block_size) || (ny % block_size) )
	{
		params->succeed = 0;
		strcpy(params->string2display,"*****ERROR: block size must divide matrix size");
		return 0;
	}


	/// INITIALIZATION
	// dx = 1 / nx ?
	dx = 1.0 / (double) (nx - 1);
	dy = 1.0 / (double) (ny - 1);


	if(params->check){
		// Set the right hand side array F.
		// This has OMP task
		rhs(nx, ny, f_, block_size);

		/*
		   Set the initial solution estimate UNEW.
		   We are "allowed" to pick up the boundary conditions exactly.
		   */
	#pragma omp parallel
	#pragma omp master
		//for collapse(2)
		for (j = 0; j < ny; j+= block_size){
			for (i = 0; i < nx; i+= block_size){
	#pragma omp task firstprivate(i,j) private(ii,jj)
				for (jj=j; jj<j+block_size; ++jj){
					for (ii=i; ii<i+block_size; ++ii)
					{
						if (ii == 0 || ii == nx - 1 || jj == 0 || jj == ny - 1) {
							(*unew)[ii][jj] = (*f)[ii][jj];
						} else {
							(*unew)[ii][jj] = 0.0;
						}
					}
				}
			}
		}
	}

	double *f_serial;
	double *u_serial;
	double *unew_serial;
	if(params->file) {
		f_serial = (double *)malloc(nx * ny * sizeof(double));
		u_serial = (double *)malloc(nx * ny * sizeof(double));
		unew_serial = (double *)malloc(nx * ny * sizeof(double));

		memcpy(f_serial,f_,nx*ny*sizeof(double));
		memcpy(u_serial,u_,nx*ny*sizeof(double));
		memcpy(unew_serial,unew_,nx*ny*sizeof(double));

		FILE* output =fopen("org.out","w");
		print2DData(output,unew_,nx,ny);
		print2DData(output,unew_,1,1);
		print2DData(output,u_,nx,ny);
		print2DData(output,u_,1,1);
		print2DData(output,f_,nx,ny);
		fclose(output);

	}


	double time;
	if(params->mode == MODE_TASK) {
		time = sweep(nx, ny, dx, dy, f_, 0, niter, u_, unew_, block_size);
	}else if(params->mode == MODE_GLOBAL) {
		time = sweep_global(nx, ny, dx, dy, f_, 0, niter, u_, unew_, block_size);
	}

	if(params->file) {
		sweep_seq(nx, ny, dx, dy, f_serial, 0, niter, u_serial, unew_serial);

		FILE* output =fopen("alg.out","w");
		fprintf(output,"unew:\n");
		print2DData(output,unew_,nx,ny);
		fprintf(output,"u:\n");
		print2DData(output,u_,nx,ny);
//		fprintf(output,"f:\n");
//		print2DData(output,f_,nx,ny);
		fclose(output);

		FILE* output2 =fopen("serial.out","w");
		fprintf(output,"unew:\n");
		print2DData(output2,unew_serial,nx,ny);
		fprintf(output,"u:\n");
		print2DData(output2,u_serial,nx,ny);
//		fprintf(output,"f:\n");
//		print2DData(output2,f_serial,nx,ny);
		fclose(output2);

	}

	if(params->check) {
		double x;
		double y;
		double *udiff_ = (double *)malloc(nx * ny * sizeof(double));
		double (*udiff)[nx][ny] = (double (*)[nx][ny])udiff_;
		/// CHECK OUTPUT
		// Check for convergence.
		for (j = 0; j < ny; j++) {
			y = (double) (j) / (double) (ny - 1);
			for (i = 0; i < nx; i++) {
				x = (double) (i) / (double) (nx - 1);
				(*udiff)[i][j] = (*unew)[i][j] - u_exact(x, y);
			}
		}
		error = r8mat_rms(nx, ny, udiff_);

		double error1;
		// Set the right hand side array F.
		rhs(nx, ny, f_, block_size);

		/*
		   Set the initial solution estimate UNEW.
		   We are "allowed" to pick up the boundary conditions exactly.
		   */
		for (j = 0; j < ny; j++) {
			for (i = 0; i < nx; i++) {
				if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1) {
					(*unew)[i][j] = (*f)[i][j];
				} else {
					(*unew)[i][j] = 0.0;
				}
			}
		}

		sweep_seq(nx, ny, dx, dy, f_, 0, niter, u_, unew_);

		// Check for convergence.
		for (j = 0; j < ny; j++) {
			y = (double) (j) / (double) (ny - 1);
			for (i = 0; i < nx; i++) {
				x = (double) (i) / (double) (nx - 1);
				(*udiff)[i][j] = (*unew)[i][j] - u_exact(x, y);
			}
		}
		error1 = r8mat_rms(nx, ny, udiff_);

		params->succeed = fabs(error - error1) < 1.0E-5;
		free(udiff_);
	}
	free(f_);
	free(u_);
	free(unew_);
	return time;
}

/* R8MAT_RMS returns the RMS norm of a vector stored as a matrix. */
double r8mat_rms(int nx, int ny, double *a_) {
	double (*a)[nx][ny] = (double (*)[nx][ny])a_;
	int i;
	int j;
	double v;

	v = 0.0;

	for (j = 0; j < ny; j++) {
		for (i = 0; i < nx; i++) {
			v = v + (*a)[i][j] * (*a)[i][j];
		}
	}
	v = sqrt(v / (double) (nx * ny));

	return v;
}

/* RHS initializes the right hand side "vector". */
void rhs(int nx, int ny, double *f_, int block_size)
{
	double (*f)[nx][ny] = (double (*)[nx][ny])f_;
	int i,ii;
	int j,jj;
	double x;
	double y;

	// The "boundary" entries of F store the boundary values of the solution.
	// The "interior" entries of F store the right hand sides of the Poisson equation.

#pragma omp parallel
#pragma omp master
	//for collapse(2)
	for (j = 0; j < ny; j+=block_size)
		for (i = 0; i < nx; i+=block_size)
#pragma omp task firstprivate(block_size,i,j,nx,ny) private(ii,jj,x,y)
			for (jj=j; jj<j+block_size; ++jj)
			{
				y = (double) (jj) / (double) (ny - 1);
				for (ii=i; ii<i+block_size; ++ii)
				{
					x = (double) (ii) / (double) (nx - 1);
					if (ii == 0 || ii == nx - 1 || jj == 0 || jj == ny - 1)
						(*f)[ii][jj] = u_exact(x, y);
					else
						(*f)[ii][jj] = - uxxyy_exact(x, y);
				}
			}
}

/* Evaluates the exact solution. */
double u_exact(double x, double y) {
	double pi = 3.141592653589793;
	double value;

	value = sin(pi * x * y);

	return value;
}

/* Evaluates (d/dx d/dx + d/dy d/dy) of the exact solution. */
double uxxyy_exact(double x, double y) {
	double pi = 3.141592653589793;
	double value;

	value = - pi * pi * (x * x + y * y) * sin(pi * x * y);

	return value;
}

#endif
